#include "hip/hip_runtime.h"
#include "FlyingEdgesAlgorithm.h"

// Function to read scalar data from file
template<typename T>
std::vector<T> readF32File(const std::string &filename, std::size_t numElements) {
    std::ifstream file(filename.c_str(), std::ios::binary);
    if (!file.is_open()) {
        throw std::runtime_error("Unable to open file: " + filename);
    }

    // Get file size
    file.seekg(0, std::ios::end);
    std::size_t fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    // Calculate the number of elements to read
    std::size_t dataSize = std::min(fileSize / sizeof(T), numElements);
    std::vector<T> data(dataSize);

    // Read data
    file.read(reinterpret_cast<char *>(data.data()), dataSize * sizeof(T));

    if (!file) {
        throw std::runtime_error("Error reading file: " + filename);
    }

    return data;
}

int main() {
    try {
        float isovalue = 8453;
        std::string filePath = "/home/exouser/dataset/SDRBENCH-EXASKY-NYX-512x512x512/temperature.f32";
        size_t numElements = 512 * 512 * 512;
        dim3 dataShape(512, 512, 512);
        std::vector<float> host_scalars;

        std::cout << "Reading file..." << std::endl;
        try {
            host_scalars = readF32File<float>(filePath, numElements);
        } catch (const std::exception &e) {
            std::cerr << "Error reading file: " << e.what() << std::endl;
            return -1;
        }
        std::cout << "File read successfully. Elements: " << host_scalars.size() << std::endl;

        // Create an instance of FlyingEdgesAlgorithm
        FlyingEdgesAlgorithm flyingEdges(host_scalars.data(), isovalue, dataShape);

        // Execute the algorithm
        flyingEdges.execute();

        // Save results to OBJ file
        // flyingEdges.saveResultsToOBJ("output.obj");
        std::cout << "Done" << std::endl;
        return 0;
    } catch (const std::exception &e) {
        std::cerr << "An error occurred: " << e.what() << std::endl;
        return -1;
    }
}
